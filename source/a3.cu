#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "matr.h"


#define THREADS_IN_BLOCK 1024


__global__ void row_normalization_gpu(float * A, int i){
  int threadId = blockIdx.y * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
  if (threadId >= N)
    return;
  float alpha = A[i * N + i];
  __syncthreads();
  A[i * N + i * threadId] /= alpha;
  return;
}


__global__ void row_elimination_gpu(float *A, int i){
  int threadId = blockIdx.y * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;
  int r = i + threadId / (N - i) + 1;
  int c = i + threadId % (N - i);
  //Check if thread should actually perform work
  int total_elements = (N - i - 1) * (N - i);
  if(threadId >= total_elements)
    return;
  A[r * N + c] = A[r * N + i] / A[i * N + i] * A[i * N + c];
  return;
}


double GE_cuda(float *B){
  float *A;

  //Allocate memory on device
  hipMalloc(&A, sizeof(float) * N * N);
  //copy the matrix from host to device
  hipMemcpy(A, (void *)B, sizeof(float) * N * N, hipMemcpyHostToDevice);

  struct timeval begin, end;
  gettimeofday(&begin, 0);

  int i=0;
  for(; i<N ; i++){
    int blocks = (int)((N - i)/(float)THREADS_IN_BLOCK);
    row_normalization_gpu<<<blocks, THREADS_IN_BLOCK>>>(A, i);
    hipDeviceSynchronize();
    blocks = (int)((N - i - 1) * (N - i)/(float)THREADS_IN_BLOCK);
    row_elimination_gpu<<<blocks, THREADS_IN_BLOCK>>>(A, i);
    hipDeviceSynchronize();
  }

  gettimeofday(&end, 0);
  double duration = (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1e-6;

  hipMemcpy(B, (void *)A, sizeof(float) * N * N, hipMemcpyDeviceToHost);
  hipFree(A);

  return duration;
}


int main(){
  float *A = create_matrix();
  float *B = create_matrix();
  initialize_matrix(A, 1, 0);
  initialize_matrix_from_another_matrix(B, A);
  double duration_gpu = GE_cuda(B);
  printf("duration was %.4f", duration);
  return 0;
}
